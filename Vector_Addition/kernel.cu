#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <cstdio>
#include <math.h>

// this kernel computes the vector sum c = a + b
// each thread performs one pair-wise addition
__global__ void vector_add(const float *a, const float *b, float *c, const size_t n) // vector_add
{
 unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
  // avoid accessing out of bounds elements
  if(i < n)
  {
    // sum elements
    c[i] = a[i] + b[i];
  }
} // end of the function vector_add

int main(void) // main function
{
  // create arrays of 1M elements
  int num_elements = 0 ;
  printf("Enter number of elements to add: ");
  scanf("%d", &num_elements);
  // compute the size of the arrays in bytes
  const int num_bytes = num_elements * sizeof(float);

  // points to host & device arrays
  float *device_array_a = 0;
  float *device_array_b = 0;
  float *device_array_c = 0;

  float *host_array_a   = 0;
  float *host_array_b   = 0;
  float *host_array_c   = 0;

  // malloc the host arrays
  host_array_a = (float*)malloc(num_bytes);
  host_array_b = (float*)malloc(num_bytes);
  host_array_c = (float*)malloc(num_bytes);

  // cudaMalloc the device arrays
 hipMalloc((void**)&device_array_a, num_bytes);
 hipMalloc((void**)&device_array_b, num_bytes);
 hipMalloc((void**)&device_array_c, num_bytes);


  // initialize host_array_a & host_array_b
  for(int i = 0; i < num_elements; ++i)
  {
    // make array a a linear ramp
    host_array_a[i] = (float)i;

    // make array b random
    host_array_b[i] = (float)rand() / RAND_MAX;
  }
    
  // copy arrays a & b to the device memory space
  hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

  vector_add <<< ceil(num_elements/32.0), 32>>>(device_array_a, device_array_b, device_array_c, num_elements);
  hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);
  for(int i = 0; i < num_elements; ++i)
  {
    printf("result %d: %1.1f + %7.1f = %7.1f\n", i, host_array_a[i], host_array_b[i], host_array_c[i]);
  }

  // deallocate memory
  free(host_array_a);
  free(host_array_b);
  free(host_array_c);

  hipFree(device_array_a);
  hipFree(device_array_b);
  hipFree(device_array_c);
  while(1);
} // end of the main function