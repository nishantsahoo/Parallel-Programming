#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void mykernel()
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	printf("Hello world from Kernel\tBlock id: %d\tThread id: %d\n", i, j);
}

int main()
{
	printf("Hello world from CPU\n");
	mykernel<<<4,5>>>();
	hipDeviceReset();
	while(1);
    return 0;
}
